#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define NUMPOINTS 10
#define ENDTIME 5
#define DT 0.1

void InitialiseToZero(float* array, int size);
__device__ void PrintPointsGPU(float* array, int size, double currentTime);
void PrintPointsCPU(float* array, int size, double currentTime);

__global__ void DiffuseHeat(float* currentPoints, float* nextPoints, int size, double dx, double dt, double endTime)
{
	unsigned int threadIndex = (threadIdx.x + blockDim.x * blockIdx.x) + 1;
	__shared__ double currentTime;
	currentTime = 0.0;
	while (currentTime < endTime)
	{
//		if (threadIndex == 1)
//		{
//			PrintPointsGPU(currentPoints, NUMPOINTS, currentTime);
//		}
		nextPoints[threadIndex] = currentPoints[threadIndex] + 0.25*(currentPoints[threadIndex+1] - (2*currentPoints[threadIndex]) + currentPoints[threadIndex-1]);
		__syncthreads();
		currentPoints[threadIndex] = nextPoints[threadIndex];
		if (threadIndex == 1)
		{
			currentTime += dt;
		}
		__syncthreads();
	}
}
int main(void)
{
	float* currentPoints = 0;
	currentPoints = (float*)malloc(NUMPOINTS*sizeof(float));
	float* nextPoints = 0;
	nextPoints = (float*)malloc(NUMPOINTS*sizeof(float));
	float* resultPoints = 0;
	resultPoints = (float*)malloc(NUMPOINTS*sizeof(float));
	float* deviceCurrentPoints = 0;
	hipMalloc((void**)&deviceCurrentPoints, NUMPOINTS*sizeof(float));
	float* deviceNextPoints = 0;
	hipMalloc((void**)&deviceNextPoints, NUMPOINTS*sizeof(float));
	if(currentPoints == 0 || nextPoints == 0 || resultPoints == 0 || deviceCurrentPoints == 0 || deviceNextPoints == 0)
	{
		printf("Couldn't allocate memory\n");
		return 1;
	}
	InitialiseToZero(currentPoints, NUMPOINTS);
	InitialiseToZero(nextPoints, NUMPOINTS);
	//make the end points some random values
	float randomValue = rand()%100;
	currentPoints[0] = randomValue;
	currentPoints[NUMPOINTS-1] = randomValue;
	hipMemcpy(deviceCurrentPoints, currentPoints, NUMPOINTS*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceNextPoints, nextPoints, NUMPOINTS*sizeof(float), hipMemcpyHostToDevice);
	const size_t blockSize = NUMPOINTS-2;
	size_t gridSize = (NUMPOINTS-2) / blockSize;
	double DX = currentPoints[1] - currentPoints[0];
	DiffuseHeat<<<gridSize, blockSize>>>(deviceCurrentPoints, deviceNextPoints, NUMPOINTS, DX, DT, ENDTIME);
	hipMemcpy(resultPoints, deviceCurrentPoints, NUMPOINTS*sizeof(float), hipMemcpyDeviceToHost);
	PrintPointsCPU(resultPoints, NUMPOINTS, ENDTIME);
	return 0;
}
void InitialiseToZero(float* array, int size)
{
	for (int index = 0; index < size; index++)
	{
		array[index] = 0;
	}
}

__device__ void PrintPointsGPU(float* array, int size, double currentTime)
{
	printf("The array values at time t=%0.1f are:\n", currentTime);
	for (int index = 0; index < size; index++)
	{
		printf("%0.2f ", array[index]);
	}
	printf("\n\n");
}

void PrintPointsCPU(float* array, int size, double currentTime)
{
	printf("The array values at time t=%0.1f are:\n", currentTime);
	for (int index = 0; index < size; index++)
	{
		printf("%0.2f ", array[index]);
	}
	printf("\n\n");
}
